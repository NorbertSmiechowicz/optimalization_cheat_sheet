#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>


__global__ void vec_conv(const float* in_vec1, const float* in_vec2, 
    size_t vec_len, float* out_vec ) 
{
    /*
    for (int a = blockIdx.x * vec_len; a < blockIdx.x * vec_len + vec_len; a++) {
        out_vec[blockIdx.x] = __fmaf_ieee_rn(in_vec1[a], in_vec2[a], out_vec[blockIdx.x]);
    }
    */
    for (int a = threadIdx.x * vec_len; a < threadIdx.x * vec_len + vec_len; a++) {
        out_vec[threadIdx.x] += in_vec1[a] * in_vec2[a];
    }
}


template<typename T>
static void print_vec(
    const std::vector<T> arr)
{
    for (int _cln = 0; _cln < arr.size(); _cln++)
    {
        std::cout << arr[_cln] << " ";
    }
    std::cout << "\n";
}

int main() {

    const size_t size = 10'000;
    const size_t thread_count = 200;

    std::vector<std::vector<float>> h_A(thread_count, std::vector<float>(size));
    std::vector<std::vector<float>> h_B(thread_count, std::vector<float>(size));
    std::vector<float> h_C(thread_count, 0);
    for (int j = 0; j < thread_count; j++) {
        for (int i = 0; i < size; i++)
        {
            h_A[j][i] = static_cast<float>(j) / 10000.0;
            h_B[j][i] = 1.0;
        }
        
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, h_A.size() * h_A[0].size() * sizeof(float));
    hipMalloc((void**)&d_B, h_B.size() * h_B[0].size() * sizeof(float));
    hipMalloc((void**)&d_C, h_C.size() * sizeof(float));

    int mem_region = h_A[0].size() * sizeof(float);
    for (int i = 0; i < h_A.size(); i++) 
    {
        hipMemcpy(d_A + i * h_A[i].size(), h_A[i].data(), mem_region, hipMemcpyHostToDevice);
        hipMemcpy(d_B + i * h_A[i].size(), h_B[i].data(), mem_region, hipMemcpyHostToDevice);
    }

    auto begin = std::chrono::steady_clock::now();

    vec_conv << <1, thread_count >> > (d_A, d_B, size, d_C);

    auto end = std::chrono::steady_clock::now();

    std::cout << "Default Time:\t\t"
        << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[ns]\n";

    hipMemcpy(h_C.data(), d_C, h_C.size() * sizeof(float), hipMemcpyDeviceToHost);

    print_vec(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}