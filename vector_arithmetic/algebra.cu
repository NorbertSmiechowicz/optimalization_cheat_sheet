#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

template<typename T>
class cuda_ptr
{
public:

    cuda_ptr() noexcept : cuda_ptr{ nullptr } {}
    explicit cuda_ptr(size_t size) noexcept : dev_size{ size }
    {
        hipError_t state = hipMalloc((void**)&dev_pointer, dev_size * sizeof(T));
        if (state == hipErrorOutOfMemory) 
        {
            dev_pointer = nullptr;
        }
    }

    ~cuda_ptr() noexcept 
    {
        hipError_t state = hipFree(dev_pointer);
    }

    size_t size() { return dev_size; }
    T* data() { return dev_pointer; }

    inline hipError_t store(std::vector<T> a)
    {
        return hipMemcpy(dev_pointer, a.data(), dev_size * sizeof(T), hipMemcpyHostToDevice);
    }

private:
    T* dev_pointer;
    size_t dev_size;
};


__global__ void vec_conv(const float* in_vec1, const float* in_vec2, 
    size_t vec_len, float* out_vec ) 
{
    /*
    for (int a = blockIdx.x * vec_len; a < blockIdx.x * vec_len + vec_len; a++) {
        out_vec[blockIdx.x] = __fmaf_ieee_rn(in_vec1[a], in_vec2[a], out_vec[blockIdx.x]);
    }
    */
    for (int a = threadIdx.x * vec_len; a < threadIdx.x * vec_len + vec_len; a++) {
        out_vec[threadIdx.x] += in_vec1[a] * in_vec2[a];
    }
}


template<typename T>
static void print_vec(
    const std::vector<T> arr)
{
    for (int _cln = 0; _cln < arr.size(); _cln++)
    {
        std::cout << arr[_cln] << " ";
    }
    std::cout << "\n";
}

int main() {

    {
        const size_t size = 10'000;
        const size_t thread_count = 200;

        std::vector<std::vector<float>> h_A(thread_count, std::vector<float>(size));
        std::vector<std::vector<float>> h_B(thread_count, std::vector<float>(size));
        std::vector<float> h_C(thread_count, 0);
        for (int j = 0; j < thread_count; j++) {
            for (int i = 0; i < size; i++)
            {
                h_A[j][i] = static_cast<float>(j) / 10000.0;
                h_B[j][i] = 1.0;
            }

        }

        float* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, h_A.size() * h_A[0].size() * sizeof(float));
        hipMalloc((void**)&d_B, h_B.size() * h_B[0].size() * sizeof(float));
        hipMalloc((void**)&d_C, h_C.size() * sizeof(float));

        int mem_region = h_A[0].size() * sizeof(float);
        for (int i = 0; i < h_A.size(); i++)
        {
            hipMemcpy(d_A + i * h_A[i].size(), h_A[i].data(), mem_region, hipMemcpyHostToDevice);
            hipMemcpy(d_B + i * h_A[i].size(), h_B[i].data(), mem_region, hipMemcpyHostToDevice);
        }

        auto begin = std::chrono::steady_clock::now();

        for (int i = 0; i < 1000; i++)
        {
            vec_conv << <1, thread_count >> > (d_A, d_B, size, d_C);
        }

        auto end = std::chrono::steady_clock::now();

        std::cout << "Default CUDA Time:\t\t\t"
            << std::chrono::duration_cast<std::chrono::microseconds> (end - begin).count() << "[us]\n";

        hipMemcpy(h_C.data(), d_C, h_C.size() * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    {
        std::cout << "\ncreating two cuda pointers\n";
        cuda_ptr<float> a(64);
        cuda_ptr<float> b(64);
        std::cout << a.data() << "\n" << b.data();
    }

    {
        std::cout << "\nnew scope\n";
        cuda_ptr<float> a(64);
        cuda_ptr<float> b(64);
        std::cout << a.data() << "\n" << b.data() << "\n\n";
     
        std::vector<float> v(10, 69.0);
        a.store(v);

        hipMemcpy(b.data(), a.data(), 40, hipMemcpyDeviceToDevice);

        std::vector<float> w(10, 0.0);
        hipMemcpy(w.data(), b.data(), 20, hipMemcpyDeviceToHost);

        for (int i = 0; i < w.size(); i++) 
        {
            std::cout << w[i] << " ";
        }
    }
    return 0;
}